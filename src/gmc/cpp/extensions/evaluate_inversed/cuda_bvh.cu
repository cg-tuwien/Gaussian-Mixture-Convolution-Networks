#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <torch/script.h>

#include <glm/glm.hpp>

#include "common.h"
#include "cuda_qt_creator_definitinos.h"
#include "lbvh/aabb.h"
#include "lbvh/bvh.h"
#include "lbvh/query.h"
#include "lbvh/predicator.h"
#include "math/symeig_cuda.h"


template<int N_DIMS, typename scalar_t>
std::ostream& operator <<(std::ostream& stream, const Gaussian<N_DIMS, scalar_t>& g) {
    stream << "Gauss[" << g.weight << "; " << g.position[0];
    for (int i = 1; i < N_DIMS; i++)
        stream << "/" << g.position[i];
    stream << "; ";

    for (int i = 0; i < N_DIMS; i++) {
        for (int j = 0; j < N_DIMS; j++) {
            if (i != 0 || j != 0)
                stream << "/";
            stream << g.covariance[i][j];
        }
    }
    stream << "]";
    return stream;
}

template <typename scalar_t, int DIMS>
__global__ void evaluate_bvh_forward(const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> mixture,
                                     const torch::PackedTensorAccessor32<lbvh::detail::Node::index_type_torch, 4, torch::RestrictPtrTraits> nodes,
                                     const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> aabbs,
                                     const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> xes,
                                     torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> sums,
                                     const gpe::MixtureAndXesNs n)
{
    using G = Gaussian<DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;
    const auto batch_index = blockIdx.x * blockDim.x + threadIdx.x;
    const auto layer_index = blockIdx.y * blockDim.y + threadIdx.y;
    const auto xes_index = blockIdx.z * blockDim.z + threadIdx.z;

    const auto batch_xes_index = min(batch_index, n.batch_xes - 1);
    const auto layer_xes_index = min(layer_index, n.layers_xes - 1);

//    printf("batch_index=%d, layer_index=%d, batch_xes_index=%d, layer_xes_index=%d, xes_index=%d\n", batch_index, layer_index, batch_xes_index, layer_xes_index, xes_index);
    if (batch_index >= n.batch || layer_index >= n.layers || xes_index >= n.xes)
        return;
//    printf("do batch_index=%d, layer_index=%d, batch_xes_index=%d, layer_xes_index=%d, xes_index=%d\n", batch_index, layer_index, batch_xes_index, layer_xes_index, xes_index);


    const unsigned int num_nodes = n.components * 2 + 1;  // (# of internal node) + (# of leaves), 2N+1
    const unsigned int num_objects = n.components;        // (# of leaves), the same as the number of objects
    const auto* bvh_nodes = &reinterpret_cast<const lbvh::detail::Node&>(nodes[batch_index][layer_index][0][0]);
    const auto* bvh_aabbs = &reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[batch_index][layer_index][0][0]);
    const auto* bvh_gaussians = &reinterpret_cast<const G&>(mixture[batch_index][layer_index][0][0]);
    Lbvh bvh {num_nodes, num_objects, bvh_nodes, bvh_aabbs, bvh_gaussians};

    const auto& x_pos = gpe::vec<DIMS>(xes[batch_xes_index][layer_xes_index][xes_index][0]);
    auto point = lbvh::make_vector_of(x_pos);
    auto& sum = sums[batch_index][layer_index][xes_index];
    auto evaluate = [bvh, &sum, &x_pos] (unsigned index) {
        const auto& g = bvh.objects[index];
        sum += gpe::evaluate_gaussian(x_pos, g.weight, g.position, g.covariance);
    };
    lbvh::query_device_with_fun(bvh, lbvh::inside_aabb(point), evaluate);
}


template <typename scalar_t, int DIMS>
__global__ void kernel_bvh_backward(const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> mixture,
                                    const torch::PackedTensorAccessor32<lbvh::detail::Node::index_type_torch, 4, torch::RestrictPtrTraits> nodes,
                                    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> aabbs,
                                    const torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> xes,
                                    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> grad_mixture,
                                    torch::PackedTensorAccessor32<scalar_t, 4, torch::RestrictPtrTraits> grad_xes,
                                    const torch::PackedTensorAccessor32<scalar_t, 3, torch::RestrictPtrTraits> grad_output,
                                    const gpe::MixtureAndXesNs n, bool requires_grad_mixture, bool requires_grad_xes)
{
    using G = Gaussian<DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;
    const auto batch_index = blockIdx.x * blockDim.x + threadIdx.x;
    const auto layer_index = blockIdx.y * blockDim.y + threadIdx.y;
    const auto xes_index = blockIdx.z * blockDim.z + threadIdx.z;

    const auto batch_xes_index = min(batch_index, n.batch_xes - 1);
    const auto layer_xes_index = min(layer_index, n.layers_xes - 1);

    if (batch_index >= n.batch || layer_index >= n.layers || xes_index >= n.xes)
        return;

    const unsigned int num_nodes = n.components * 2 + 1;  // (# of internal node) + (# of leaves), 2N+1
    const unsigned int num_objects = n.components;        // (# of leaves), the same as the number of objects
    const auto* bvh_nodes = &reinterpret_cast<const lbvh::detail::Node&>(nodes[batch_index][layer_index][0][0]);
    const auto* bvh_aabbs = &reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[batch_index][layer_index][0][0]);
    const auto* bvh_gaussians = &reinterpret_cast<const G&>(mixture[batch_index][layer_index][0][0]);
    Lbvh bvh {num_nodes, num_objects, bvh_nodes, bvh_aabbs, bvh_gaussians};

    const auto& x_pos = gpe::vec<DIMS>(xes[batch_xes_index][layer_xes_index][xes_index][0]);
    auto point = lbvh::make_vector_of<scalar_t>(x_pos);

    auto current_grad_mixture = grad_mixture[batch_index][layer_index];
    auto current_grad_xes = grad_xes[batch_xes_index][layer_xes_index][xes_index];
    const auto current_grad_output = grad_output[batch_index][layer_index][xes_index];

    auto evaluate_backward = [&] (unsigned index) {
        const G& g = bvh.objects[index];

        const auto t = x_pos - g.position;
        const auto v = scalar_t(-0.5) * glm::dot(t, (g.covariance * t));
        const auto exp = gpe::exp(v);
        const auto weighted_exp = g.weight * exp;
        const auto local_grad_c_pos = weighted_exp * t * g.covariance;

        if (requires_grad_xes) {
            const auto grad_xes_addition = - current_grad_output * local_grad_c_pos;
            for (uint i = 0; i < DIMS; ++i) {
                atomicAdd(&current_grad_xes[i], grad_xes_addition[i]);
            }
        }
        if (requires_grad_mixture) {
            const auto grad_c_weight_addition = exp * current_grad_output;
            const auto grad_c_pos_addition = local_grad_c_pos * current_grad_output;
            const auto grad_c_cov_addition = - g.weight * scalar_t(0.5) * exp * current_grad_output * glm::outerProduct(t, t);
            atomicAdd(&current_grad_mixture[index][0], grad_c_weight_addition);
            for (uint i = 0; i < DIMS; ++i) {
                atomicAdd(&current_grad_mixture[index][1 + i], grad_c_pos_addition[i]);
                for (uint j = 0; j < DIMS; ++j)
                    atomicAdd(&current_grad_mixture[index][1 + DIMS + i*DIMS + j], grad_c_cov_addition[i][j]);
            }
        }

    };
    lbvh::query_device_with_fun(bvh, lbvh::inside_aabb(point), evaluate_backward);
}


torch::Tensor inverse_permutation(const torch::Tensor& p) {
    auto l = torch::arange(p.size(-1), torch::TensorOptions(p.device()).dtype(p.dtype()));
    auto shape = p.sizes().vec();
    assert(shape.size() > 0);
    std::for_each(shape.begin(), shape.end() - 1, [](auto& i) { i = 1; });
    l = l.view(shape).expand_as(p);
    return torch::scatter(torch::empty_like(p), -1, p, l);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> cuda_bvh_forward_impl(const at::Tensor& mixture, const at::Tensor& xes) {
    using namespace torch::indexing;
    using LBVH = lbvh::bvh<float, Gaussian<2, float>>;

    auto n = gpe::check_input_and_get_ns(mixture, xes);
    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor");
    TORCH_CHECK(n.batch * n.layers < 65535, "n_batch x n_layers must be smaller than 65535 for CUDA");
    TORCH_CHECK(n.xes < 65535, "number of xes must be smaller than 65535 for CUDA");
    TORCH_CHECK(n.components > 1, "number of components must be greater 1 for this implementation");
    TORCH_CHECK(n.dims == 2, "atm only 2d gaussians");
    TORCH_CHECK(mixture.dtype() == caffe2::TypeMeta::Make<float>(), "atm only float");

    auto bvh = LBVH(mixture);
    torch::Tensor sum = torch::zeros({n.batch, n.layers, n.xes}, torch::dtype(mixture.dtype()).device(mixture.device()));

    // mixture(batch, layer, component, data)
    // xes(batch, layer, n, data)

    auto xes_copy = xes;
    const auto use_indirect_xes = n.xes == n.components && n.batch == n.batch_xes && n.layers == n.layers_xes;
    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        indices = indices.view({n.batch, n.layers, n.components, 1}).expand_as(xes);
        xes_copy = torch::gather(xes, 2, indices);
    }

    dim3 dimBlock = dim3(1, 1, LBVH_N_QUERY_THREADS);
    dim3 dimGrid = dim3((n.batch + dimBlock.x - 1) / dimBlock.x,
                        (n.layers + dimBlock.y - 1) / dimBlock.y,
                        (n.xes + dimBlock.z - 1) / dimBlock.z);
//    printf("dimBlock=(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
//    printf("dimGrid=(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);


//    auto start = std::chrono::high_resolution_clock::now();

    AT_DISPATCH_FLOATING_TYPES(mixture.scalar_type(), "cuda_bvh_backward_impl", ([&] {
        auto sum_a = sum.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();
        auto mixture_a = mixture.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        auto nodes_a = bvh.m_nodes.packed_accessor32<lbvh::detail::Node::index_type_torch, 4, torch::RestrictPtrTraits>();
        auto aabbs_a = bvh.m_aabbs.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        const auto xes_a = xes_copy.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();

        if (n.dims == 2)
            evaluate_bvh_forward<scalar_t, 2><<<dimGrid, dimBlock>>>(mixture_a, nodes_a, aabbs_a, xes_a, sum_a, n);
        else
            evaluate_bvh_forward<scalar_t, 3><<<dimGrid, dimBlock>>>(mixture_a, nodes_a, aabbs_a, xes_a, sum_a, n);
    }));

//    hipDeviceSynchronize();
//    auto end = std::chrono::high_resolution_clock::now();
//    std::cout << "bvh eval elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count() << "ms\n";

    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        indices = inverse_permutation(indices);
        sum = torch::gather(sum, 2, indices);
    }

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    return std::make_tuple(sum, bvh.m_nodes, bvh.m_aabbs);
}

std::tuple<torch::Tensor, torch::Tensor> cuda_bvh_backward_impl(const torch::Tensor& grad_output,
                                                  const torch::Tensor& mixture, const torch::Tensor& bvh_nodes, const torch::Tensor& aabbs,
                                                  const torch::Tensor& xes,
                                                  bool requires_grad_mixture, bool requires_grad_xes) {
    using namespace torch::indexing;
    using LBVH = lbvh::bvh<float, Gaussian<2, float>>;
    gpe::check_mixture(mixture);
    auto n = gpe::check_input_and_get_ns(mixture, xes);

    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor")
    TORCH_CHECK(grad_output.device().is_cuda(), "grad_output must be a CUDA tensor");
    TORCH_CHECK(grad_output.dim() == 3, "grad_output has wrong number of dimensions");
    TORCH_CHECK(grad_output.size(0) == n.batch, "grad_output has wrong batch dimension");
    TORCH_CHECK(grad_output.size(1) == n.layers, "grad_output has wrong layer dimension");
    TORCH_CHECK(grad_output.size(2) == n.xes, "grad_output has wrong xes dimension");
    TORCH_CHECK(grad_output.dtype() == mixture.dtype(), "grad_output dtype does not match with mixture dtype")

    auto bvh = LBVH(mixture, bvh_nodes, aabbs);
    torch::Tensor grad_mixture = torch::zeros_like(mixture);
    torch::Tensor grad_xes = torch::zeros_like(xes);

    dim3 dimBlock = dim3(1, 1, LBVH_N_QUERY_THREADS);
    dim3 dimGrid = dim3((n.batch + dimBlock.x - 1) / dimBlock.x,
                        (n.layers + dimBlock.y - 1) / dimBlock.y,
                        (n.xes + dimBlock.z - 1) / dimBlock.z);

    auto xes_copy = xes;
    auto grad_output_copy = grad_output;
    const auto use_indirect_xes = n.xes == n.components && n.batch == n.batch_xes && n.layers == n.layers_xes;
    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        xes_copy = torch::gather(xes, 2, indices.view({n.batch_xes, n.layers_xes, n.xes, 1}).expand_as(xes));
        grad_output_copy = torch::gather(grad_output, 2, indices);
    }


    AT_DISPATCH_FLOATING_TYPES(mixture.scalar_type(), "cuda_bvh_backward_impl", ([&] {
        auto mixture_a = mixture.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        auto nodes_a = bvh.m_nodes.packed_accessor32<lbvh::detail::Node::index_type_torch, 4, torch::RestrictPtrTraits>();
        auto aabbs_a = bvh.m_aabbs.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        auto xes_a = xes_copy.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        auto grad_mixture_a = grad_mixture.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        auto grad_xes_a = grad_xes.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
        auto grad_output_a = grad_output_copy.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

        if (n.dims == 2)
            kernel_bvh_backward<scalar_t, 2><<<dimGrid, dimBlock>>>(mixture_a, nodes_a, aabbs_a, xes_a,
                                                                    grad_mixture_a, grad_xes_a, grad_output_a,
                                                                    n, requires_grad_mixture, requires_grad_xes);
        else
            kernel_bvh_backward<scalar_t, 3><<<dimGrid, dimBlock>>>(mixture_a, nodes_a, aabbs_a, xes_a,
                                                                    grad_mixture_a, grad_xes_a, grad_output_a,
                                                                    n, requires_grad_mixture, requires_grad_xes);
    }));

    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        indices = inverse_permutation(indices);
        grad_xes = torch::gather(grad_xes, 2, indices.view({n.batch_xes, n.layers_xes, n.xes, 1}).expand_as(xes));
    }
    return {grad_mixture, grad_xes};
}
