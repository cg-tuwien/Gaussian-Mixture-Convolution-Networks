#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <torch/script.h>
#include <torch/nn/functional.h>

#include <glm/glm.hpp>

#include "common.h"
#include "lbvh/aabb.h"
#include "lbvh/bvh.h"
#include "lbvh/query.h"
#include "lbvh/predicator.h"
#include "math/symeig.h"

#ifndef __HIPCC__
constexpr dim3 blockIdx;
constexpr dim3 blockDim;
constexpr dim3 threadIdx;
using std::min;
using std::max;

namespace torch {
template <typename T>
struct RestrictPtrTraits {
  typedef T* __restrict__ PtrType;
};
}

#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// __device__ code can't do constexpr variables
#define GPE_BVH_BUFFER_SIZE 100u

template<int N_DIMS, typename scalar_t>
struct Gaussian {
    scalar_t weight;
    glm::vec<N_DIMS, scalar_t> position;
    glm::mat<N_DIMS, N_DIMS, scalar_t> covariance;
};
template<int N_DIMS, typename scalar_t>
std::ostream& operator <<(std::ostream& stream, const Gaussian<N_DIMS, scalar_t>& g) {
    stream << "Gauss[" << g.weight << "; " << g.position[0];
    for (int i = 1; i < N_DIMS; i++)
        stream << "/" << g.position[i];
    stream << "; ";

    for (int i = 0; i < N_DIMS; i++) {
        for (int j = 0; j < N_DIMS; j++) {
            if (i != 0 || j != 0)
                stream << "/";
            stream << g.covariance[i][j];
        }
    }
    stream << "]";
    return stream;
}

torch::Tensor cuda_bvh_forward_impl(const at::Tensor& mixture, const at::Tensor& xes) {
    using namespace torch::indexing;
    namespace F = torch::nn::functional;
    using LBVH = lbvh::bvh<float, Gaussian<2, float>>;

    auto start = std::chrono::steady_clock::now();
    auto n = gpe::check_input_and_get_ns(mixture, xes);

    torch::Tensor sum = torch::zeros({n.batch, n.layers, n.xes}, torch::dtype(mixture.dtype()).device(mixture.device()));
//    const auto xes_a = xes.packed_accessor32<float, 4, torch::RestrictPtrTraits>();
//    auto sum_a = sum.packed_accessor32<float, 3, torch::RestrictPtrTraits>();

    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor");
    TORCH_CHECK(n.batch * n.layers < 65535, "n_batch x n_layers must be smaller than 65535 for CUDA");
    TORCH_CHECK(n.xes < 65535, "number of xes must be smaller than 65535 for CUDA");
    TORCH_CHECK(n.components > 1, "number of components must be greater 1 for this implementation");

    TORCH_CHECK(n.dims == 2, "atm only 2d gaussians");
    TORCH_CHECK(mixture.dtype() == caffe2::TypeMeta::Make<float>(), "atm only float");

    torch::Tensor aabbs;
    {
        constexpr float threshold = 0.0001f;

        torch::Tensor factors = -2 * torch::log(threshold / torch::abs(gpe::weights(mixture)));
        factors = factors.where(factors > 0, torch::zeros({1, 1, 1}, factors.device()));
        factors = torch::sqrt(factors);

        torch::Tensor covs = gpe::covariances(mixture).inverse();
        torch::Tensor eigenvalues;
        torch::Tensor eigenvectors;

        std::tie(eigenvalues, eigenvectors) = gpe::symeig(covs);
        /*
         * eigenvectors is a tensor of [*, *, *, d, d], where d is the dimensionality (2 or 3)
         * the eigenvectors are in the rows of that d * d matrix.
         */
        eigenvalues = torch::sqrt(eigenvalues);
        eigenvectors = eigenvalues.unsqueeze(-1) * eigenvectors;

        auto ellipsoidM = factors.unsqueeze(-1).unsqueeze(-1) * eigenvectors;

        // https://stackoverflow.com/a/24112864/4032670
        // https://members.loria.fr/SHornus/ellipsoid-bbox.html
        // we take the norm over the eigenvectors, that is analogous to simon fraiss' code in gmvis/core/Gaussian.cpp
        auto delta = torch::norm(ellipsoidM, 2, {-2});
        auto centroid = gpe::positions(mixture);
        auto upper = centroid + delta;
        auto lower = centroid - delta;

        // bring that thing into a format that can be read by our lbvh builder
        upper = F::pad(upper, F::PadFuncOptions({0, 4-n.dims}));
        lower = F::pad(lower, F::PadFuncOptions({0, 4-n.dims}));
        aabbs = torch::cat({upper, lower}, -1).contiguous();
    }

    for (int batch_id = 0; batch_id < int(n.batch); ++batch_id) {
        for (int layer_id = 0; layer_id < int(n.layers); ++layer_id) {
            torch::Tensor current_mixture = mixture.index({batch_id, layer_id});
            TORCH_CHECK(current_mixture.is_contiguous(), "mixtures must be contiguous");
            auto mixture_begin = static_cast<Gaussian<2, float>*>(current_mixture.data_ptr());
            auto mixture_end = mixture_begin + n.components;
            torch::Tensor current_aabbs = aabbs.index({batch_id, layer_id});
            auto aabbs_begin = static_cast<lbvh::aabb<float>*>(current_aabbs.data_ptr());

            auto bvh = LBVH(mixture_begin, mixture_end, aabbs_begin);
            auto batch_id_xes = std::min(batch_id, int(n.batch_xes)-1);
            auto layer_id_xes = std::min(layer_id, int(n.layers_xes)-1);
            const auto current_xes = xes.index({batch_id_xes, layer_id_xes});
            auto current_sum = sum.index({batch_id, layer_id});
            const auto xes_a = current_xes.packed_accessor32<float, 2, torch::RestrictPtrTraits>();
            auto sum_a = current_sum.packed_accessor32<float, 1, torch::RestrictPtrTraits>();

            const auto bvh_dev = bvh.get_device_repr();
            thrust::for_each(thrust::device,
                thrust::make_counting_iterator<std::size_t>(0),
                thrust::make_counting_iterator<std::size_t>(n.xes),
                [bvh_dev, xes_a, sum_a, n] __device__ (std::size_t idx) mutable {
                    const float& v = xes_a[idx][0];
                    const auto& x_pos = gpe::vec<2>(v);
                    unsigned int buffer[GPE_BVH_BUFFER_SIZE];
                    auto point = float4{x_pos.x, x_pos.y, 0, 0};
                    const auto num_found = lbvh::query_device(bvh_dev, lbvh::inside_aabb(point), buffer, GPE_BVH_BUFFER_SIZE);
                    for (int i = 0; i < min(GPE_BVH_BUFFER_SIZE, num_found); i++) {
                        const auto& g = bvh_dev.objects[buffer[i]];
                        sum_a[idx] += gpe::evaluate_gaussian(x_pos, g.weight, g.position, g.covariance);
                    }
                    return ;
                });
        }
    }


    gpuErrchk(hipDeviceSynchronize());
    auto end = std::chrono::steady_clock::now();
    std::cout << "elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count() << "ms\n";
    return sum;
}

//std::vector<torch::Tensor> cuda_parallel_backward_impl(torch::Tensor grad_output, torch::Tensor mixture, torch::Tensor xes, bool requires_grad_mixture, bool requires_grad_xes) {
//    gpe::check_mixture(mixture);
//    auto n = gpe::check_input_and_get_ns(mixture, xes);

//    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor")
//    TORCH_CHECK(grad_output.device().is_cuda(), "grad_output must be a CUDA tensor");
//    TORCH_CHECK(grad_output.dim() == 3, "grad_output has wrong number of dimensions");
//    TORCH_CHECK(grad_output.size(0) == n.batch, "grad_output has wrong batch dimension");
//    TORCH_CHECK(grad_output.size(1) == n.layers, "grad_output has wrong layer dimension");
//    TORCH_CHECK(grad_output.size(2) == n.xes, "grad_output has wrong xes dimension");
//    TORCH_CHECK(grad_output.dtype() == mixture.dtype(), "grad_output dtype does not match with mixture dtype")


//    torch::Tensor grad_mixture = torch::zeros({n.batch, n.layers, n.components, mixture.size(3)}, torch::dtype(mixture.dtype()).device(mixture.device()));
//    torch::Tensor grad_xes = torch::zeros({n.batch_xes, n.layers_xes, n.xes, n.dims}, torch::dtype(mixture.dtype()).device(mixture.device()));

//    dim3 dimBlock = dim3(128);
//    const dim3 dimGrid = dim3(n.batch * n.layers,
//                              n.xes,
//                              (n.components + dimBlock.z - 1) / dimBlock.z);
////    std::cout << "forward: dimBlock=" << dimBlock.x << "/" << dimBlock.y << "/" << dimBlock.z << ", dimGrid=" << dimGrid.x << "/" << dimGrid.y << "/" << dimGrid.z << std::endl;

//    AT_DISPATCH_FLOATING_TYPES(mixture.scalar_type(), "eval_inversed_omp_backward", ([&] {
//        auto mixture_a = mixture.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
//        auto xes_a = xes.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
//        auto grad_mixture_a = grad_mixture.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
//        auto grad_xes_a = grad_xes.packed_accessor32<scalar_t, 4, torch::RestrictPtrTraits>();
//        auto grad_output_a = grad_output.packed_accessor32<scalar_t, 3, torch::RestrictPtrTraits>();

//        if (n.dims == 2)
//            kernel_backward<scalar_t, 2><<<dimGrid, dimBlock>>>(mixture_a, xes_a, grad_mixture_a, grad_xes_a, grad_output_a, n, requires_grad_mixture, requires_grad_xes);
//        else
//            kernel_backward<scalar_t, 3><<<dimGrid, dimBlock>>>(mixture_a, xes_a, grad_mixture_a, grad_xes_a, grad_output_a, n, requires_grad_mixture, requires_grad_xes);
//    }));

//    return {grad_mixture, grad_xes};
//}
