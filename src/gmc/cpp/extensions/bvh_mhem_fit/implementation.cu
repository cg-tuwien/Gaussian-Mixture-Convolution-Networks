#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <torch/script.h>

#include <glm/glm.hpp>

#include "common.h"
#include "mixture.h"
#include "cuda_qt_creator_definitinos.h"
#include "lbvh/aabb.h"
#include "lbvh/bvh.h"
#include "lbvh/query.h"
#include "lbvh/predicator.h"
#include "math/symeig_cuda.h"


#include "bvh_mhem_fit/implementation.h"
#include "parallel_start.h"

namespace bvh_mhem_fit {

namespace  {

template<int N_DIMS, typename scalar_t>
std::ostream& operator <<(std::ostream& stream, const Gaussian<N_DIMS, scalar_t>& g) {
    stream << "Gauss[" << g.weight << "; " << g.position[0];
    for (int i = 1; i < N_DIMS; i++)
        stream << "/" << g.position[i];
    stream << "; ";

    for (int i = 0; i < N_DIMS; i++) {
        for (int j = 0; j < N_DIMS; j++) {
            if (i != 0 || j != 0)
                stream << "/";
            stream << g.covariance[i][j];
        }
    }
    stream << "]";
    return stream;
}

template <typename scalar_t, int DIMS, template <typename U> class PtrTraits = gpe::RestrictPtrTraits>
__host__ __device__
void evaluate_bvh_forward(const dim3& gpe_gridDim, const dim3& gpe_blockDim,
                         const dim3& gpe_blockIdx, const dim3& gpe_threadIdx,
                         const torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> mixture,
                         const torch::PackedTensorAccessor32<lbvh::detail::Node::index_type_torch, 4, PtrTraits> nodes,
                         const torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> aabbs,
                         const torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> xes,
                         torch::PackedTensorAccessor32<scalar_t, 3, PtrTraits> sums,
                         const gpe::MixtureAndXesNs n)
{
    GPE_UNUSED(gpe_gridDim)
    using G = Gaussian<DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;
    const int batch_index = int(gpe_blockIdx.x * gpe_blockDim.x + gpe_threadIdx.x);
    const int layer_index = int(gpe_blockIdx.y * gpe_blockDim.y + gpe_threadIdx.y);
    const int xes_index = int(gpe_blockIdx.z * gpe_blockDim.z + gpe_threadIdx.z);

    const auto batch_xes_index = min(batch_index, int(n.batch_xes - 1));
    const auto layer_xes_index = min(layer_index, int(n.layers_xes - 1));

    //    printf("batch_index=%d, layer_index=%d, batch_xes_index=%d, layer_xes_index=%d, xes_index=%d\n", batch_index, layer_index, batch_xes_index, layer_xes_index, xes_index);
    if (batch_index >= int(n.batch) || layer_index >= int(n.layers) || xes_index >= int(n.xes))
        return;
    //    printf("do batch_index=%d, layer_index=%d, batch_xes_index=%d, layer_xes_index=%d, xes_index=%d\n", batch_index, layer_index, batch_xes_index, layer_xes_index, xes_index);


    const unsigned int num_nodes = n.components * 2 + 1;  // (# of internal node) + (# of leaves), 2N+1
    const unsigned int num_objects = n.components;        // (# of leaves), the same as the number of objects
    const auto* bvh_nodes = &reinterpret_cast<const lbvh::detail::Node&>(nodes[batch_index][layer_index][0][0]);
    const auto* bvh_aabbs = &reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[batch_index][layer_index][0][0]);
    const auto* bvh_gaussians = &reinterpret_cast<const G&>(mixture[batch_index][layer_index][0][0]);
    Lbvh bvh {num_nodes, num_objects, bvh_nodes, bvh_aabbs, bvh_gaussians};

    const auto& x_pos = gpe::vec<DIMS>(xes[batch_xes_index][layer_xes_index][xes_index][0]);
    auto point = lbvh::make_vector_of(x_pos);
    auto& sum = sums[batch_index][layer_index][xes_index];
    auto evaluate = [bvh, &sum, &x_pos] (unsigned index) {
        const auto& g = bvh.objects[index];
        sum += gpe::evaluate_gaussian(x_pos, g.weight, g.position, g.covariance);
    };
    lbvh::query_device_with_fun(bvh, lbvh::inside_aabb(point), evaluate);
}

template <typename scalar_t, int DIMS, template <typename U> class PtrTraits = gpe::RestrictPtrTraits>
__host__ __device__
void kernel_bvh_backward(const dim3& gpe_blockIdx, const dim3& gpe_blockDim,
                        const dim3& gpe_threadIdx, const dim3& gpe_threadDim,
                        const torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> mixture,
                        const torch::PackedTensorAccessor32<lbvh::detail::Node::index_type_torch, 4, PtrTraits> nodes,
                        const torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> aabbs,
                        const torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> xes,
                        torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> grad_mixture,
                        torch::PackedTensorAccessor32<scalar_t, 4, PtrTraits> grad_xes,
                        const torch::PackedTensorAccessor32<scalar_t, 3, PtrTraits> grad_output,
                        const gpe::MixtureAndXesNs n, bool requires_grad_mixture, bool requires_grad_xes)
{
    GPE_UNUSED(gpe_threadDim)
    using G = Gaussian<DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;
    const int batch_index = int(gpe_blockIdx.x * gpe_blockDim.x + gpe_threadIdx.x);
    const int layer_index = int(gpe_blockIdx.y * gpe_blockDim.y + gpe_threadIdx.y);
    const int xes_index = int(gpe_blockIdx.z * gpe_blockDim.z + gpe_threadIdx.z);

    const auto batch_xes_index = min(batch_index, int(n.batch_xes - 1));
    const auto layer_xes_index = min(layer_index, int(n.layers_xes - 1));

    if (batch_index >= int(n.batch) || layer_index >= int(n.layers) || xes_index >= int(n.xes))
        return;
    printf("do batch_index=%d, layer_index=%d, batch_xes_index=%d, layer_xes_index=%d, xes_index=%d\n", batch_index, layer_index, batch_xes_index, layer_xes_index, xes_index);

    const unsigned int num_nodes = n.components * 2 + 1;  // (# of internal node) + (# of leaves), 2N+1
    const unsigned int num_objects = n.components;        // (# of leaves), the same as the number of objects
    const auto* bvh_nodes = &reinterpret_cast<const lbvh::detail::Node&>(nodes[batch_index][layer_index][0][0]);
    const auto* bvh_aabbs = &reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[batch_index][layer_index][0][0]);
    const auto* bvh_gaussians = &reinterpret_cast<const G&>(mixture[batch_index][layer_index][0][0]);
    Lbvh bvh {num_nodes, num_objects, bvh_nodes, bvh_aabbs, bvh_gaussians};

    const auto& x_pos = gpe::vec<DIMS>(xes[batch_xes_index][layer_xes_index][xes_index][0]);
    auto point = lbvh::make_vector_of<scalar_t>(x_pos);

    auto current_grad_mixture = grad_mixture[batch_index][layer_index];
    auto current_grad_xes = grad_xes[batch_xes_index][layer_xes_index][xes_index];
    const auto current_grad_output = grad_output[batch_index][layer_index][xes_index];

    auto evaluate_backward = [&] (int index) {
        const G& g = bvh.objects[index];

        const auto t = x_pos - g.position;
        const auto v = scalar_t(-0.5) * glm::dot(t, (g.covariance * t));
        const auto exp = gpe::exp(v);
        const auto weighted_exp = g.weight * exp;
        const auto local_grad_c_pos = weighted_exp * t * g.covariance;

        if (requires_grad_xes) {
            const auto grad_xes_addition = - current_grad_output * local_grad_c_pos;
            for (int i = 0; i < DIMS; ++i) {
                gpe::atomicAdd(&current_grad_xes[i], grad_xes_addition[i]);
            }
        }
        if (requires_grad_mixture) {
            const auto grad_c_weight_addition = exp * current_grad_output;
            const auto grad_c_pos_addition = local_grad_c_pos * current_grad_output;
            const auto grad_c_cov_addition = - g.weight * scalar_t(0.5) * exp * current_grad_output * glm::outerProduct(t, t);
            gpe::atomicAdd(&current_grad_mixture[index][0], grad_c_weight_addition);
            for (int i = 0; i < DIMS; ++i) {
                gpe::atomicAdd(&current_grad_mixture[index][1 + i], grad_c_pos_addition[i]);
                for (int j = 0; j < DIMS; ++j)
                    gpe::atomicAdd(&current_grad_mixture[index][1 + DIMS + i*DIMS + j], grad_c_cov_addition[i][j]);
            }
        }

    };
    lbvh::query_device_with_fun(bvh, lbvh::inside_aabb(point), evaluate_backward);
}


torch::Tensor inverse_permutation(const torch::Tensor& p) {
    auto l = torch::arange(p.size(-1), torch::TensorOptions(p.device()).dtype(p.dtype()));
    auto shape = p.sizes().vec();
    assert(shape.size() > 0);
    std::for_each(shape.begin(), shape.end() - 1, [](auto& i) { i = 1; });
    l = l.view(shape).expand_as(p);
    return torch::scatter(torch::empty_like(p), -1, p, l);
}

} // anonymous namespace

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> cuda_bvh_forward_impl(const at::Tensor& mixture, const at::Tensor& xes) {
    using namespace torch::indexing;
    using LBVH = lbvh::bvh<float, Gaussian<2, float>>;

    auto n = gpe::check_input_and_get_ns(mixture, xes);
    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor")
    TORCH_CHECK(n.batch * n.layers < 65535, "n_batch x n_layers must be smaller than 65535 for CUDA")
    TORCH_CHECK(n.xes < 65535, "number of xes must be smaller than 65535 for CUDA")
    TORCH_CHECK(n.components > 1, "number of components must be greater 1 for this implementation")
    TORCH_CHECK(n.dims == 2, "atm only 2d gaussians")
    TORCH_CHECK(mixture.dtype() == caffe2::TypeMeta::Make<float>(), "atm only float")

    auto bvh = LBVH(mixture);
    torch::Tensor sum = torch::zeros({n.batch, n.layers, n.xes}, torch::dtype(mixture.dtype()).device(mixture.device()));

    // mixture(batch, layer, component, data)
    // xes(batch, layer, n, data)

    auto xes_copy = xes;
    const auto use_indirect_xes = n.xes == n.components && n.batch == n.batch_xes && n.layers == n.layers_xes;
    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        indices = indices.view({n.batch, n.layers, n.components, 1}).expand_as(xes);
        xes_copy = torch::gather(xes, 2, indices);
    }

    dim3 dimBlock = dim3(1, 1, LBVH_N_QUERY_THREADS);
    dim3 dimGrid = dim3((n.batch + dimBlock.x - 1) / dimBlock.x,
                        (n.layers + dimBlock.y - 1) / dimBlock.y,
                        (n.xes + dimBlock.z - 1) / dimBlock.z);
    //    printf("dimBlock=(%d, %d, %d)\n", dimBlock.x, dimBlock.y, dimBlock.z);
    //    printf("dimGrid=(%d, %d, %d)\n", dimGrid.x, dimGrid.y, dimGrid.z);


    //    auto start = std::chrono::high_resolution_clock::now();


    AT_DISPATCH_FLOATING_TYPES(mixture.scalar_type(), "cuda_bvh_backward_impl", ([&] {
                                   auto sum_a = sum.packed_accessor32<scalar_t, 3, gpe::RestrictPtrTraits>();
                                   auto mixture_a = mixture.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   auto nodes_a = bvh.m_nodes.packed_accessor32<lbvh::detail::Node::index_type_torch, 4, gpe::RestrictPtrTraits>();
                                   auto aabbs_a = bvh.m_aabbs.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   const auto xes_a = xes_copy.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();

                                   if (n.dims == 2) {
                                       auto fun = [mixture_a, nodes_a, aabbs_a, xes_a, sum_a, n] __host__ __device__
                                           (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                               evaluate_bvh_forward<scalar_t, 2>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx, mixture_a, nodes_a, aabbs_a, xes_a, sum_a, n);
                                           };
                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                   }
                                   else {
                                       auto fun = [mixture_a, nodes_a, aabbs_a, xes_a, sum_a, n] __host__ __device__
                                           (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                               evaluate_bvh_forward<scalar_t, 3>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx, mixture_a, nodes_a, aabbs_a, xes_a, sum_a, n);
                                           };
                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                   }
                               }));

    //    hipDeviceSynchronize();
    //    auto end = std::chrono::high_resolution_clock::now();
    //    std::cout << "bvh eval elapsed time: " << std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count() << "ms\n";

    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        indices = inverse_permutation(indices);
        sum = torch::gather(sum, 2, indices);
    }

    GPE_CUDA_ASSERT(hipPeekAtLastError())
    GPE_CUDA_ASSERT(hipDeviceSynchronize())

    return std::make_tuple(sum, bvh.m_nodes, bvh.m_aabbs);
}

std::tuple<torch::Tensor, torch::Tensor> cuda_bvh_backward_impl(const torch::Tensor& grad_output,
                                                                const torch::Tensor& mixture, const torch::Tensor& bvh_nodes, const torch::Tensor& aabbs,
                                                                const torch::Tensor& xes,
                                                                bool requires_grad_mixture, bool requires_grad_xes) {
    using namespace torch::indexing;
    using LBVH = lbvh::bvh<float, Gaussian<2, float>>;
    gpe::check_mixture(mixture);
    auto n = gpe::check_input_and_get_ns(mixture, xes);

    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor")
    TORCH_CHECK(grad_output.device().is_cuda(), "grad_output must be a CUDA tensor")
    TORCH_CHECK(grad_output.dim() == 3, "grad_output has wrong number of dimensions")
    TORCH_CHECK(grad_output.size(0) == n.batch, "grad_output has wrong batch dimension")
    TORCH_CHECK(grad_output.size(1) == n.layers, "grad_output has wrong layer dimension")
    TORCH_CHECK(grad_output.size(2) == n.xes, "grad_output has wrong xes dimension")
    TORCH_CHECK(grad_output.dtype() == mixture.dtype(), "grad_output dtype does not match with mixture dtype")

    auto bvh = LBVH(mixture, bvh_nodes, aabbs);
    torch::Tensor grad_mixture = torch::zeros_like(mixture);
    torch::Tensor grad_xes = torch::zeros_like(xes);

    dim3 dimBlock = dim3(1, 1, LBVH_N_QUERY_THREADS);
    dim3 dimGrid = dim3((n.batch + dimBlock.x - 1) / dimBlock.x,
                        (n.layers + dimBlock.y - 1) / dimBlock.y,
                        (n.xes + dimBlock.z - 1) / dimBlock.z);

    auto xes_copy = xes;
    auto grad_output_copy = grad_output;
    const auto use_indirect_xes = n.xes == n.components && n.batch == n.batch_xes && n.layers == n.layers_xes;
    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        xes_copy = torch::gather(xes, 2, indices.view({n.batch_xes, n.layers_xes, n.xes, 1}).expand_as(xes));
        grad_output_copy = torch::gather(grad_output, 2, indices);
    }


    AT_DISPATCH_FLOATING_TYPES(mixture.scalar_type(), "cuda_bvh_backward_impl", ([&] {
                                   auto mixture_a = mixture.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   auto nodes_a = bvh.m_nodes.packed_accessor32<lbvh::detail::Node::index_type_torch, 4, gpe::RestrictPtrTraits>();
                                   auto aabbs_a = bvh.m_aabbs.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   auto xes_a = xes_copy.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   auto grad_mixture_a = grad_mixture.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   auto grad_xes_a = grad_xes.packed_accessor32<scalar_t, 4, gpe::RestrictPtrTraits>();
                                   auto grad_output_a = grad_output_copy.packed_accessor32<scalar_t, 3, gpe::RestrictPtrTraits>();

                                   if (n.dims == 2) {
                                       auto fun = [=] __host__ __device__
                                           (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                               kernel_bvh_backward<scalar_t, 2>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx,
                                                                                mixture_a, nodes_a, aabbs_a, xes_a,
                                                                                grad_mixture_a, grad_xes_a, grad_output_a,
                                                                                n, requires_grad_mixture, requires_grad_xes);
                                           };
                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                   }
                                   else {
                                       auto fun = [=] __host__ __device__
                                           (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                               kernel_bvh_backward<scalar_t, 3>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx,
                                                                                mixture_a, nodes_a, aabbs_a, xes_a,
                                                                                grad_mixture_a, grad_xes_a, grad_output_a,
                                                                                n, requires_grad_mixture, requires_grad_xes);
                                           };
                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                   }
                               }));

    if (use_indirect_xes) {
        auto indices = bvh.m_nodes.index({Slice(), Slice(), Slice(bvh.m_n_internal_nodes, None), 3}).to(torch::ScalarType::Long);
        indices = inverse_permutation(indices);
        grad_xes = torch::gather(grad_xes, 2, indices.view({n.batch_xes, n.layers_xes, n.xes, 1}).expand_as(xes));
    }
    return {grad_mixture, grad_xes};
}

} // namespace bvh_mhem_fit
