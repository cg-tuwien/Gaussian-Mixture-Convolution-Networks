#include "hip/hip_runtime.h"
#include "bvh_mhem_fit/implementation.h"
#include <algorithm>
#include <chrono>
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <glm/matrix.hpp>
#include <torch/types.h>

#include "bvh_mhem_fit/implementation_common.cuh"
#include "common.h"
#include "cuda_qt_creator_definitinos.h"
#include "hacked_accessor.h"
#include "lbvh/aabb.h"
#include "lbvh/bvh.h"
#include "lbvh/query.h"
#include "lbvh/predicator.h"
#include "math/symeig_cuda.h"
#include "mixture.h"
#include "parallel_start.h"

namespace bvh_mhem_fit {

namespace  {


template <typename scalar_t, int DIMS>
__host__ __device__ void iterate_over_nodes(const dim3& gpe_gridDim, const dim3& gpe_blockDim,
                                                const dim3& gpe_blockIdx, const dim3& gpe_threadIdx,
                                                const gpe::PackedTensorAccessor32<scalar_t, 3> mixture,
                                                const gpe::PackedTensorAccessor32<lbvh::detail::Node::index_type_torch, 3> nodes,
                                                const gpe::PackedTensorAccessor32<scalar_t, 3> aabbs,
                                                torch::PackedTensorAccessor32<int, 2> flags,
                                                const gpe::MixtureNs n, const unsigned n_mixtures, const unsigned n_internal_nodes, const unsigned n_nodes)
{
    GPE_UNUSED(gpe_gridDim)
    using G = gpe::Gaussian<DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;

    const auto node_id = gpe_blockIdx.x * gpe_blockDim.x + gpe_threadIdx.x + n_internal_nodes;
    const auto mixture_id = gpe_blockIdx.y * gpe_blockDim.y + gpe_threadIdx.y;
    if (mixture_id >= n_mixtures || node_id >= n_nodes)
        return;

    const auto* node = &reinterpret_cast<const lbvh::detail::Node&>(nodes[int(mixture_id)][int(node_id)][0]);
    while(node->parent_idx != lbvh::detail::Node::index_type(0xFFFFFFFF)) // means idx == 0
    {
        auto* flag = &reinterpret_cast<int&>(flags[mixture_id][node->parent_idx]);
        const int old = gpe::atomicCAS(flag, 0, 1);
        if(old == 0)
        {
            // this is the first thread entered here.
            // wait the other thread from the other child node.
            return;
        }
        assert(old == 1);
        // here, the flag has already been 1. it means that this
        // thread is the 2nd thread. merge AABB of both childlen.


        auto& current_aabb = reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[mixture_id][node->parent_idx][0]);
        node = &reinterpret_cast<const lbvh::detail::Node&>(nodes[mixture_id][node->parent_idx][0]);
        const auto& left_aabb = reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[mixture_id][node->left_idx][0]);
        const auto& right_aabb = reinterpret_cast<const lbvh::Aabb<scalar_t>&>(aabbs[mixture_id][node->right_idx][0]);
    }
}

} // anonymous namespace

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> forward_impl(const at::Tensor& mixture, int n_components_target) {
    using namespace torch::indexing;
    using LBVH = lbvh::Bvh<float, gpe::Gaussian<2, float>>;

    // todo: flatten mixture for kernel, i.g. nbatch/nlayers/ncomponents/7 => nmixture/ncomponents/7

    auto n = gpe::get_ns(mixture);
    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor")
    TORCH_CHECK(n.batch * n.layers < 65535, "n_batch x n_layers must be smaller than 65535 for CUDA")
    TORCH_CHECK(n.components > 1, "number of components must be greater 1 for this implementation")
    TORCH_CHECK(n.dims == 2, "atm only 2d gaussians")
    TORCH_CHECK(mixture.dtype() == caffe2::TypeMeta::Make<float>(), "atm only float")

    auto bvh = LBVH(mixture);
    auto n_mixtures = n.batch * n.layers;
    auto n_internal_nodes = bvh.m_n_internal_nodes;
    auto n_nodes = bvh.m_n_nodes;
    auto flag_container = torch::zeros({n_mixtures, bvh.m_n_internal_nodes}, torch::TensorOptions(mixture.device()).dtype(torch::ScalarType::Int));
    const auto flags_a = flag_container.packed_accessor32<int, 2>();

    dim3 dimBlock = dim3(32, 1, 1);
    dim3 dimGrid = dim3((uint(bvh.m_n_leaf_nodes) + dimBlock.x - 1) / dimBlock.x,
                        (uint(n.layers) + dimBlock.y - 1) / dimBlock.y,
                        (uint(n.batch) + dimBlock.z - 1) / dimBlock.z);

    auto mixture_c = mixture.cpu();
    auto bvh_nodes_c = bvh.m_nodes.cpu();
    auto bvh_aabbs_c = bvh.m_aabbs.cpu();

    AT_DISPATCH_FLOATING_TYPES(mixture.scalar_type(), "cuda_bvh_backward_impl", ([&] {
//                                   auto mixture_a = gpe::accessor<scalar_t, 3>(mixture);
//                                   auto nodes_a = gpe::accessor<lbvh::detail::Node::index_type_torch, 3>(bvh.m_nodes);
//                                   auto aabbs_a = gpe::accessor<scalar_t, 3>(bvh.m_aabbs);
                                   auto mixture_a = gpe::accessor<scalar_t, 3>(mixture_c);
                                   auto nodes_a = gpe::accessor<lbvh::detail::Node::index_type_torch, 3>(bvh_nodes_c);
                                   auto aabbs_a = gpe::accessor<scalar_t, 3>(bvh_aabbs_c);

                                   if (n.dims == 2) {
                                       auto fun = [mixture_a, nodes_a, aabbs_a, flags_a, n, n_mixtures, n_internal_nodes, n_nodes] __host__ __device__
                                           (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                               iterate_over_nodes<scalar_t, 2>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx,
                                                                               mixture_a, nodes_a, aabbs_a, flags_a,
                                                                               n, n_mixtures, n_internal_nodes, n_nodes);
                                           };
//                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture_c), dimGrid, dimBlock, fun);
                                   }
                                   else {
                                       auto fun = [mixture_a, nodes_a, aabbs_a, flags_a, n, n_mixtures, n_internal_nodes, n_nodes] __host__ __device__
                                           (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                               iterate_over_nodes<scalar_t, 3>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx,
                                                                               mixture_a, nodes_a, aabbs_a, flags_a,
                                                                               n, n_mixtures, n_internal_nodes, n_nodes);
                                           };
//                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                       gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture_c), dimGrid, dimBlock, fun);
                                   }
                               }));

    GPE_CUDA_ASSERT(hipPeekAtLastError())
    GPE_CUDA_ASSERT(hipDeviceSynchronize())

//    return std::make_tuple(sum, bvh.m_nodes, bvh.m_aabbs);
    //todo: return something useful
    return {};
}


} // namespace bvh_mhem_fit
