#include "hip/hip_runtime.h"
#include "bvh_mhem_fit/implementation.h"
#include <algorithm>
#include <chrono>
#include <vector>
#include <stdio.h>

#include <hip/hip_runtime.h>
//#include <
#include <hip/hip_runtime.h>
#include <glm/matrix.hpp>
#include <torch/types.h>

#include "bvh_mhem_fit/implementation_common.cuh"
#include "common.h"
#include "cuda_qt_creator_definitinos.h"
#include "cuda_operations.h"
#include "hacked_accessor.h"
#include "lbvh/aabb.h"
#include "lbvh/bvh.h"
#include "lbvh/query.h"
#include "lbvh/predicator.h"
#include "math/symeig_cuda.h"
#include "mixture.h"
#include "parallel_start.h"

namespace bvh_mhem_fit {

namespace  {

using node_index_torch_t = lbvh::detail::Node::index_type_torch;
using node_index_t = lbvh::detail::Node::index_type;


template <int REDUCTION_N>
__host__ __device__ int copy_gaussian_ids(gpe::Accessor32<node_index_torch_t, 1> tmp_g_container_source, node_index_torch_t* destination) {
    for (int i = 0; i < REDUCTION_N; ++i) {
        destination[i] = tmp_g_container_source[i];
        if (tmp_g_container_source[i] == -1)
            return i;
    }
    return REDUCTION_N;
}


template <int REDUCTION_N>
__host__ __device__ int collect_child_gaussian_ids(const lbvh::detail::Node* node,
                                                   gpe::Accessor32<node_index_torch_t, 2> tmp_g_container_a,
                                                   node_index_torch_t* destination) {
    auto n_copied = copy_gaussian_ids<REDUCTION_N>(tmp_g_container_a[node->left_idx], destination);
    destination += n_copied;
    n_copied += copy_gaussian_ids<REDUCTION_N>(tmp_g_container_a[node->right_idx], destination);
    return n_copied;
}


template <typename scalar_t, int N_DIMS, int REDUCTION_N>
__host__ __device__ void fit_reduce_node(const lbvh::detail::Node* node,
                                         gpe::Accessor32<node_index_torch_t, 2> tmp_g_container_a,
                                         gpe::Accessor32<scalar_t, 2> mixture_a) {
    // for now (testing) simply select N_GAUSSIANS_TARGET strongest gaussians
    // no stl available in cuda 10.1.
    node_index_torch_t gaussian_ids[REDUCTION_N * 2];
    scalar_t gaussian_v[REDUCTION_N * 2];
    const auto n_input_gaussians = collect_child_gaussian_ids<REDUCTION_N>(node, tmp_g_container_a, gaussian_ids);
    int largest_index = -1;
    scalar_t largest_value = 0;
    for (int i = 0; i < n_input_gaussians; ++i) {
        gaussian_v[i] = gpe::gaussian<N_DIMS>(mixture_a[gaussian_ids[i]]).weight;
        if (gaussian_v[i] > largest_value) {
            largest_value = gaussian_v[i];
            largest_index = i;
        }
    }

    assert(largest_index != -1);
    int currently_largest_i = 0;
    tmp_g_container_a[node->object_idx][currently_largest_i] = node_index_torch_t(largest_index);
    for (int i = 1; i < REDUCTION_N; ++i) {
        scalar_t currently_largest_v = 0;
        currently_largest_i = -1;
        for (int j = 0; j < n_input_gaussians; ++j) {
            auto v = gaussian_v[j];
            if (v > currently_largest_v && (v < largest_value || (v == largest_value && j > largest_index))) {
                currently_largest_v = v;
                currently_largest_i = j;
            }
        }

        tmp_g_container_a[node->object_idx][i] = node_index_torch_t(currently_largest_i);
        largest_value = currently_largest_v;
        largest_index = currently_largest_i;
    }
}


template <int REDUCTION_N>
__host__ __device__ int count_gaussians(const lbvh::detail::Node* node,
                                        gpe::Accessor32<node_index_torch_t, 2> tmp_g_container_a) {
    auto c = tmp_g_container_a[node->object_idx];
    for (int i = 0; i < REDUCTION_N; ++i) {
        if (c[i] == -1)
            return i;
    }
    return REDUCTION_N;
}


template <int N_GAUSSIANS_TARGET>
__host__ __device__ int count_child_gaussians(const lbvh::detail::Node* node,
                                               const gpe::Accessor32<node_index_torch_t, 2>& nodes_a,
                                               gpe::Accessor32<node_index_torch_t, 2> tmp_g_container_a) {
    return count_gaussians<N_GAUSSIANS_TARGET>(reinterpret_cast<const lbvh::detail::Node*>(&nodes_a[node->left_idx][0]), tmp_g_container_a)
           + count_gaussians<N_GAUSSIANS_TARGET>(reinterpret_cast<const lbvh::detail::Node*>(&nodes_a[node->right_idx][0]), tmp_g_container_a);
}


template <typename scalar_t, int N_DIMS, int REDUCTION_N>
__host__ __device__ void iterate_over_nodes(const dim3& gpe_gridDim, const dim3& gpe_blockDim,
                                            const dim3& gpe_blockIdx, const dim3& gpe_threadIdx,
                                            gpe::PackedTensorAccessor32<scalar_t, 3> mixture,
                                            const gpe::PackedTensorAccessor32<node_index_torch_t, 3> nodes,
                                            const gpe::PackedTensorAccessor32<scalar_t, 3> aabbs,
                                            gpe::PackedTensorAccessor32<int, 2> flags,
                                            gpe::PackedTensorAccessor32<node_index_torch_t, 3> tmp_g_container_a,
                                            const gpe::MixtureNs n, const int n_mixtures, const unsigned n_internal_nodes, const unsigned n_nodes, int n_components_target)
{
    GPE_UNUSED(gpe_gridDim)
    using G = gpe::Gaussian<N_DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;

    auto node_id = node_index_t(gpe_blockIdx.x * gpe_blockDim.x + gpe_threadIdx.x + n_internal_nodes);
    const auto mixture_id = int(gpe_blockIdx.y * gpe_blockDim.y + gpe_threadIdx.y);
    if (mixture_id >= n_mixtures || node_id >= n_nodes)
        return;

    // collect Gs int tmp_g_container until N_GAUSSIANS_TARGET * 2 is reached
    // then merge and cont


    const auto* node = reinterpret_cast<const lbvh::detail::Node*>(&nodes[mixture_id][int(node_id)][0]);
    tmp_g_container_a[mixture_id][node_id][0] = node_index_torch_t(node->object_idx);
    while(node->parent_idx != node_index_t(0xFFFFFFFF)) // means idx == 0
    {
        auto* flag = &reinterpret_cast<int&>(flags[mixture_id][node->parent_idx]);
        const int old = gpe::atomicCAS(flag, 0, 1);
        if(old == 0)
        {
            // this is the first thread entered here.
            // wait the other thread from the other child node.
            return;
        }
        assert(old == 1);
        // here, the flag has already been 1. it means that this
        // thread is the 2nd thread. merge AABB of both childlen.

        node_id = node->parent_idx;
        node = reinterpret_cast<const lbvh::detail::Node*>(&nodes[mixture_id][node_id][0]);
        if (count_child_gaussians<REDUCTION_N>(node, nodes[mixture_id], tmp_g_container_a[mixture_id]) > REDUCTION_N) {
            fit_reduce_node<scalar_t, N_DIMS, REDUCTION_N>(node, tmp_g_container_a[mixture_id], mixture[mixture_id]);
        }
        else {
            auto* destination = &tmp_g_container_a[mixture_id][node->object_idx][0];
            collect_child_gaussian_ids<REDUCTION_N>(node, tmp_g_container_a[mixture_id], destination);
        }
    }
}

template <typename scalar_t, int N_DIMS, int REDUCTION_N>
__host__ __device__ void collect_result(const dim3& gpe_gridDim, const dim3& gpe_blockDim,
                                            const dim3& gpe_blockIdx, const dim3& gpe_threadIdx,
                                            const gpe::PackedTensorAccessor32<scalar_t, 3> mixture,
                                            gpe::PackedTensorAccessor32<scalar_t, 3> out_mixture,
                                            const gpe::PackedTensorAccessor32<node_index_torch_t, 3> nodes,
                                            const gpe::PackedTensorAccessor32<scalar_t, 3> aabbs,
                                            gpe::PackedTensorAccessor32<int, 2> flags,
                                            gpe::PackedTensorAccessor32<node_index_torch_t, 3> tmp_g_container_a,
                                            const gpe::MixtureNs n, const int n_mixtures, const unsigned n_internal_nodes, const unsigned n_nodes, int n_components_target)
{
    GPE_UNUSED(gpe_gridDim)
    using G = gpe::Gaussian<N_DIMS, scalar_t>;
    using Lbvh = lbvh::detail::basic_device_bvh<scalar_t, G, true>;

    auto target_component_id = node_index_t(gpe_blockIdx.x * gpe_blockDim.x + gpe_threadIdx.x + n_internal_nodes);
    const auto mixture_id = int(gpe_blockIdx.y * gpe_blockDim.y + gpe_threadIdx.y);
    if (mixture_id >= n_mixtures || target_component_id >= n_components_target)
        return;

    // collect merged Gs
    auto n_levels_down = 32 - gpe::clz(uint32_t(n_components_target / REDUCTION_N));

    const auto* node = reinterpret_cast<const lbvh::detail::Node*>(&nodes[mixture_id][int(0)][0]);

    //todo cathch: that'll fail if the tree is very unbalanced -> abort and pad gaussians with zeroes;
    for (int i = 0; i < n_levels_down; ++i) {
        auto decision_bit = 1 << (n_levels_down - i);
        if (target_component_id & decision_bit)
            node = node = reinterpret_cast<const lbvh::detail::Node*>(&nodes[mixture_id][node->left_idx][0]);
        else
            node = node = reinterpret_cast<const lbvh::detail::Node*>(&nodes[mixture_id][node->right_idx][0]);
    }
}


} // anonymous namespace


std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> forward_impl(at::Tensor mixture, int n_components_target = 32) {
    using namespace torch::indexing;
    using LBVH = lbvh::Bvh<2, float>;

    constexpr int N_GAUSSIANS_TARGET = 4;

    // todo: flatten mixture for kernel, i.g. nbatch/nlayers/ncomponents/7 => nmixture/ncomponents/7

    auto n = gpe::get_ns(mixture);
    TORCH_CHECK(mixture.device().is_cuda(), "mixture must be a CUDA tensor")
    TORCH_CHECK(n.batch * n.layers < 65535, "n_batch x n_layers must be smaller than 65535 for CUDA")
    TORCH_CHECK(n.components > 1, "number of components must be greater 1 for this implementation")
    TORCH_CHECK(n.dims == 2, "atm only 2d gaussians")
    TORCH_CHECK(mixture.dtype() == caffe2::TypeMeta::Make<float>(), "atm only float")

    const auto n_mixtures = n.batch * n.layers;
    const auto bvh = LBVH(mixture);
    mixture = mixture.view({n_mixtures, n.components, -1});
    auto scratch_mixture = mixture.clone();
    const auto n_internal_nodes = bvh.m_n_internal_nodes;
    const auto n_nodes = bvh.m_n_nodes;
    auto flag_container = torch::zeros({n_mixtures, n_internal_nodes}, torch::TensorOptions(mixture.device()).dtype(torch::ScalarType::Int));
    auto tmp_g_container = -1 * torch::ones({n_mixtures, n_nodes, N_GAUSSIANS_TARGET},
                                            torch::TensorOptions(mixture.device()).dtype(lbvh::detail::TorchTypeMapper<node_index_torch_t>::id()));
    auto flags_a = gpe::accessor<int, 2>(flag_container);
    auto tmp_g_container_a = gpe::accessor<node_index_torch_t, 3>(tmp_g_container);


    GPE_DISPATCH_FLOATING_TYPES_AND_DIM(mixture.scalar_type(), n.dims, ([&] {
                                    dim3 dimBlock = dim3(32, 1, 1);
                                    dim3 dimGrid = dim3((uint(bvh.m_n_leaf_nodes) + dimBlock.x - 1) / dimBlock.x,
                                                        (uint(n_mixtures) + dimBlock.y - 1) / dimBlock.y,
                                                        (uint(1) + dimBlock.z - 1) / dimBlock.z);

                                   auto mixture_a = gpe::accessor<scalar_t, 3>(scratch_mixture);
                                   auto nodes_a = gpe::accessor<lbvh::detail::Node::index_type_torch, 3>(bvh.m_nodes);
                                   auto aabbs_a = gpe::accessor<scalar_t, 3>(bvh.m_aabbs);

                                   auto fun = [mixture_a, nodes_a, aabbs_a, flags_a, tmp_g_container_a, n, n_mixtures, n_internal_nodes, n_nodes, n_components_target] __host__ __device__
                                       (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                           iterate_over_nodes<scalar_t, N_DIMS, 4>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx,
                                                                                   mixture_a, nodes_a, aabbs_a, flags_a, tmp_g_container_a,
                                                                                   n, n_mixtures, n_internal_nodes, n_nodes, n_components_target);
                                       };
                                   gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                   GPE_CUDA_ASSERT(hipPeekAtLastError())
                                   GPE_CUDA_ASSERT(hipDeviceSynchronize())
                               }));


    auto out_mixture = torch::zeros({n_mixtures, n.components, mixture.size(-1)}, torch::TensorOptions(mixture.device()).dtype(mixture.dtype()));
    GPE_DISPATCH_FLOATING_TYPES_AND_DIM(mixture.scalar_type(), n.dims, ([&] {
                                            dim3 dimBlock = dim3(32, 1, 1);
                                            dim3 dimGrid = dim3((uint(n_components_target) + dimBlock.x - 1) / dimBlock.x,
                                                                (uint(n_mixtures) + dimBlock.y - 1) / dimBlock.y,
                                                                (uint(1) + dimBlock.z - 1) / dimBlock.z);

                                            auto mixture_a = gpe::accessor<scalar_t, 3>(scratch_mixture);
                                            auto out_mixture_a = gpe::accessor<scalar_t, 3>(out_mixture);
                                            auto nodes_a = gpe::accessor<lbvh::detail::Node::index_type_torch, 3>(bvh.m_nodes);
                                            auto aabbs_a = gpe::accessor<scalar_t, 3>(bvh.m_aabbs);

                                            auto fun = [mixture_a, out_mixture_a, nodes_a, aabbs_a, flags_a, tmp_g_container_a, n, n_mixtures, n_internal_nodes, n_nodes, n_components_target]
                                                __host__ __device__
                                                (const dim3& gpe_gridDim, const dim3& gpe_blockDim, const dim3& gpe_blockIdx, const dim3& gpe_threadIdx) {
                                                    collect_result<scalar_t, N_DIMS, 4>(gpe_gridDim, gpe_blockDim, gpe_blockIdx, gpe_threadIdx,
                                                                                        mixture_a, out_mixture_a, nodes_a, aabbs_a, flags_a, tmp_g_container_a,
                                                                                        n, n_mixtures, n_internal_nodes, n_nodes, n_components_target);
                                                };
                                            gpe::start_parallel<gpe::ComputeDevice::Both>(gpe::device(mixture), dimGrid, dimBlock, fun);
                                        }));

    GPE_CUDA_ASSERT(hipPeekAtLastError())
    GPE_CUDA_ASSERT(hipDeviceSynchronize())

    return std::make_tuple(out_mixture.view({n.batch, n.layers, n.components, -1}), bvh.m_nodes, bvh.m_aabbs);
}


} // namespace bvh_mhem_fit

